
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <time.h>



// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
#include <math.h>
void API_add_v2(float *a, float *b, float *c, int data_num)
{
    printf("In function: data nums = %d\n",data_num);

    // Number of threads in each thread block
    int blockSize = 1024;
 
    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)data_num/blockSize);
    vecAdd<<<gridSize, blockSize>>>(a, b, c, data_num);
}