
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd3(double *a, double *b, double *c,double *d,int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        d[id] = a[id] + b[id]+c[id];
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd4(double *a, double *b, double *c,double *d,double *e,int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        e[id] = a[id] + b[id]+c[id]+d[id];
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd5(double *a, double *b, double *c,double *d,double *e,double *f,int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        f[id] = a[id] + b[id]+c[id]+d[id]+e[id];
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 100000;

    int tensor_num=0;
    int tensor_size=0;

    clock_t start1,start2, end2,end1;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
    double *h_d;
    double *h_e;
    double *h_f;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
    double *d_d;
    double *d_e;
    double *d_f;
    {
       tensor_num=atoi(argv[1]);
       tensor_size=atoi(argv[2]);
       n=tensor_size;
       //printf("Tensor num: %d, tensor_size: %d\n",tensor_num,tensor_size);
    }
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
    h_d = (double*)malloc(bytes);
    h_e = (double*)malloc(bytes);
    h_f = (double*)malloc(bytes);
    
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    hipMalloc(&d_d, bytes);
    hipMalloc(&d_e, bytes);
    hipMalloc(&d_f, bytes);
 
    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }  

    start1=clock();
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
    if(tensor_num>=3)
        hipMemcpy( d_c, h_c, bytes, hipMemcpyHostToDevice);
    if(tensor_num>=4)
        hipMemcpy( d_d, h_d, bytes, hipMemcpyHostToDevice);
    if(tensor_num>=5)
        hipMemcpy( d_e, h_e, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
hipDeviceSynchronize();
    start2=clock();
 
    // Execute the kernel
    if(tensor_num==2)
        vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    if(tensor_num==3)
        vecAdd3<<<gridSize, blockSize>>>(d_a, d_b, d_c,d_d, n);
    if(tensor_num==4)
        vecAdd4<<<gridSize, blockSize>>>(d_a, d_b, d_c,d_d,d_e, n);
    if(tensor_num==5)
        vecAdd5<<<gridSize, blockSize>>>(d_a, d_b, d_c,d_d,d_e,d_f, n);
    hipDeviceSynchronize();
    end2=clock();
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
    end1=clock();
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    //for(i=0; i<n; i++)
    //    sum += h_c[i];
    //printf("final result: %f\n", sum/n);

    float time1 = (float)(end1 - start1) / CLOCKS_PER_SEC;
    float time2 = (float)(end2 - start2) / CLOCKS_PER_SEC;
    printf("[%d, %d]: %f ms, computing: %f ms\n", tensor_num,tensor_size, time1*1000, time2*1000);
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_e);
    hipFree(d_f);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_d);
    free(h_e);
    free(h_f);
 
    return 0;
}